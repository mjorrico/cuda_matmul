#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "matrixfunctions.hpp"
#include "run_kernel.cuh"

#include "kernels/001_matmul_naive.cuh"
#include "kernels/002_matmul_coalesced.cuh"
#include "kernels/003_matmul_coalesced_v2.cuh"
#include "kernels/004_matmul_cacheblock.cuh"
#include "kernels/005_matmul_cacheblock_v2.cuh"
#include "kernels/006_matmul_blocktiling1d.cuh"

void mmul_benchmark(mmulFunc mmul, float* dev_a, float* dev_b, float* dev_c, float* c, int N, double gflop, double memoryio) {
    float elapsed_time, best_time = 1e9;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < 10; i++) {
        hipEventRecord(start);

        mmul(dev_a, dev_b, dev_c, N);

        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        best_time = imin(best_time, elapsed_time);
    }

    std::cout << "Throughput: " << 1e3 * gflop / best_time << " GFLOPs." << std::endl; // GLOP / ms
    std::cout << "Bandwidth: " << memoryio / best_time << " GB/s." << std::endl << std::endl; // MB / ms

    hipMemcpy(c, dev_c, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void run_mmul_naive(float* a, float* b, float* c, int N) {
    dim3 gridsize((N + 31) / 32, (N + 31) / 32);
    dim3 blocksize(32, 32);

    mmul_naive<<<gridsize, blocksize>>>(a, b, c, N);
}

void run_mmul_coalesced(float* a, float* b, float* c, int N) {
    dim3 gridsize((N + 31) / 32, (N + 31) / 32);
    dim3 blocksize(32 * 32);

    mmul_coalesced<<<gridsize, blocksize>>>(a, b, c, N);
}

void run_mmul_coalesced_v2(float* a, float* b, float* c, int N) {
    dim3 gridsize((N + 31) / 32, (N + 31) / 32);
    dim3 blocksize(32, 32);

    mmul_coalesced_v2<<<gridsize, blocksize>>>(a, b, c, N);
}

void run_mmul_cacheblock(float* a, float* b, float* c, int N) {
    dim3 gridsize((N + 31) / 32, (N + 31) / 32);
    dim3 blocksize(32 * 32);

    mmul_cacheblock<<<gridsize, blocksize>>>(a, b, c, N);
}

void run_mmul_cacheblock_v2(float* a, float* b, float* c, int N) {
    dim3 gridsize((N + 31) / 32, (N + 31) / 32);
    dim3 blocksize(32, 32);

    mmul_cacheblock_v2<<<gridsize, blocksize>>>(a, b, c, N);
}

void run_mmul_blocktiling1d(float* a, float* b, float* c, int N) {
    const uint BM = 64;
    const uint BK = 8;
    const uint BN = 64;
    const uint TM = 8;

    dim3 gridsize((N + BM - 1) / BM, (N + BN - 1) / BN);
    dim3 blocksize(BM * BN / TM);

    assert(BM == BN);
    assert(BK == BM / TM);

    mmul_blocktiling1d<BM, BN, BK, TM><<<gridsize, blocksize>>>(a, b, c, N);
}
