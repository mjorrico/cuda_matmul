#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void mmul_coalesced_v2(float* a, float* b, float* c, int N) {
    // swapped so that memory access to A is coalesced
    const uint col = blockIdx.x * blockDim.x + threadIdx.x;
    const uint row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N) {
        float temp = 0;
        for (int i = 0; i < N; i++) {
            temp += a[row * N + i] * b[i * N + col];
        }

        c[row * N + col] = temp;
    }
}